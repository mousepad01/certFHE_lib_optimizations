#include "hip/hip_runtime.h"
#include "gpu_processing_support.h"

#if CERTFHE_USE_CUDA

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <iostream>

const int MAX_BLOCK_PER_GRID_COUNT = 65535;
const int MAX_THREADS_PER_BLOCK = 1024;

/**
 * Device function
 * Each thread operates on default length chunks
**/
__global__ void ctxt_multiply_kernel(uint64_t deflen_to_uint64, uint64_t result_deflen_cnt, uint64_t snd_deflen_cnt,
	uint64_t * result, const uint64_t * fst, const uint64_t * snd) {

	int result_deflen_offset = blockDim.x * blockIdx.x + threadIdx.x;
	int result_deflen_stride = blockDim.x * gridDim.x;

	for (int result_deflen_i = result_deflen_offset; result_deflen_i < result_deflen_cnt; result_deflen_i += result_deflen_stride) {

		int fst_deflen_i = (result_deflen_i / snd_deflen_cnt) * deflen_to_uint64;
		int snd_deflen_i = (result_deflen_i % snd_deflen_cnt) * deflen_to_uint64;

		for (int i = 0; i < deflen_to_uint64; i++)
			result[i + result_deflen_i * deflen_to_uint64] = fst[i + fst_deflen_i] & snd[i + snd_deflen_i];
	}
}

__global__ void ctxt_decrypt_kernel(uint64_t deflen_to_uint64, uint64_t to_decrypt_deflen_cnt, const uint64_t * to_decrypt, const uint64_t * sk_mask,
									int * decryption_result) {

	int to_decrypt_deflen_offset = blockDim.x * blockIdx.x + threadIdx.x;
	int to_decrypt_deflen_stride = blockDim.x * gridDim.x;

	int local_decryption_result = 1;

	for (int to_decrypt_deflen_i = to_decrypt_deflen_offset; to_decrypt_deflen_i < to_decrypt_deflen_cnt; to_decrypt_deflen_i += to_decrypt_deflen_stride) {

		for (int i = 0; i < deflen_to_uint64; i++)
			local_decryption_result &= ((to_decrypt[to_decrypt_deflen_i * deflen_to_uint64 + i] & sk_mask[i]) ^ sk_mask[i]) == (uint64_t)0;
		
		(void)atomicXor(decryption_result, local_decryption_result);
	}
}

/**
 * called from CCC class, linked with extern specifier
 * receives as argument the WHOLE ciphertexts
**/
__host__ void CUDA_chiphertext_multiply(uint64_t deflen_to_uint64, uint64_t result_deflen_cnt, uint64_t fst_deflen_cnt, uint64_t snd_deflen_cnt,
	uint64_t * result, const uint64_t * fst, const uint64_t * snd) {

	uint64_t * VRAM_result;
	uint64_t * VRAM_fst;
	uint64_t * VRAM_snd;

	hipMalloc(&VRAM_result, (uint64_t)result_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));
	hipMalloc(&VRAM_fst, (uint64_t)fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));
	hipMalloc(&VRAM_snd, (uint64_t)snd_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));

	hipMemcpy(VRAM_result, result, (uint64_t)result_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(VRAM_fst, fst, (uint64_t)fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(VRAM_snd, snd, (uint64_t)snd_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), hipMemcpyHostToDevice);

	int threads_per_block = result_deflen_cnt > MAX_THREADS_PER_BLOCK ? MAX_THREADS_PER_BLOCK : (int)result_deflen_cnt;

	int block_cnt = (int)(result_deflen_cnt / MAX_THREADS_PER_BLOCK);
	if (result_deflen_cnt % MAX_THREADS_PER_BLOCK)
		block_cnt += 1;

	ctxt_multiply_kernel <<< block_cnt, threads_per_block >>> (deflen_to_uint64, result_deflen_cnt, snd_deflen_cnt, VRAM_result, VRAM_fst, VRAM_snd);
	hipDeviceSynchronize();

	hipMemcpy(result, VRAM_result, (uint64_t)result_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), hipMemcpyDeviceToHost);

	hipFree(VRAM_result);
	hipFree(VRAM_fst);
	hipFree(VRAM_snd);
}

__host__ int CUDA_ciphertext_decrpytion(uint64_t deflen_to_uint64, uint64_t to_decrypt_deflen_cnt, const uint64_t * to_decrypt, const uint64_t * sk_mask) {

	uint64_t * VRAM_to_decrypt;
	uint64_t * VRAM_sk_mask;

	int * VRAM_decryption_result;

	hipMalloc(&VRAM_to_decrypt, to_decrypt_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));
	hipMalloc(&VRAM_sk_mask, deflen_to_uint64 * sizeof(uint64_t));

	hipMalloc(&VRAM_decryption_result, sizeof(int));

	hipMemcpy(VRAM_to_decrypt, to_decrypt, to_decrypt_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(VRAM_sk_mask, sk_mask, deflen_to_uint64 * sizeof(uint64_t), hipMemcpyHostToDevice);

	int threads_per_block = to_decrypt_deflen_cnt > MAX_THREADS_PER_BLOCK ? MAX_THREADS_PER_BLOCK : (int)to_decrypt_deflen_cnt;

	int block_cnt = (int)(to_decrypt_deflen_cnt / MAX_THREADS_PER_BLOCK);
	if (to_decrypt_deflen_cnt % MAX_THREADS_PER_BLOCK)
		block_cnt += 1;

	ctxt_decrypt_kernel <<< block_cnt, threads_per_block >>> (deflen_to_uint64, to_decrypt_deflen_cnt, VRAM_to_decrypt, VRAM_sk_mask, VRAM_decryption_result);
	hipDeviceSynchronize();

	int decryption_result;

	hipMemcpy(&decryption_result, VRAM_decryption_result, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(VRAM_to_decrypt);
	hipFree(VRAM_sk_mask);
	hipFree(VRAM_decryption_result);

	return decryption_result;
}

#endif